#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include ""

#include <iostream>
#include <cassert>
#include <vector>
#include <algorithm>

#include "constants.h"

using namespace std;

hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG) || defined(CUDA_DEBUG)
	if (result != hipSuccess)
	{
		cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << endl;
		assert(result == hipSuccess);
	}
#endif
	return result;
}

__global__ void gpu_filter(unsigned char* src_img, unsigned char* dst_img, int width) {
	
	uint2 offset = {
		blockIdx.x * THREADS_X + threadIdx.x,
		blockIdx.y * THREADS_Y + threadIdx.y
	};
	
	char neighbors[9];
	neighbors[0] = src_img[offset.y * width + offset.x];
	neighbors[1] = src_img[offset.y * width + offset.x + 1];
	neighbors[2] = src_img[offset.y * width + offset.x + 2];
	neighbors[3] = src_img[(offset.y + 1) * width + offset.x];
	neighbors[4] = src_img[(offset.y + 1) * width + offset.x + 1];
	neighbors[5] = src_img[(offset.y + 1) * width + offset.x + 2];
	neighbors[6] = src_img[(offset.y + 2) * width + offset.x];
	neighbors[7] = src_img[(offset.y + 2) * width + offset.x + 1];
	neighbors[8] = src_img[(offset.y + 2) * width + offset.x + 2];

	char min = neighbors[0];
	for (int i = 1; i < 9; i++) {
		if (neighbors[i] < min)
			min = neighbors[i];
	}

	dst_img[(offset.y + 1) * width + offset.x + 1] = min;
}