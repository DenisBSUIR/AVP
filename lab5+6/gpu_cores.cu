#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include ""

#include <iostream>
#include <cassert>
#include <vector>
#include <algorithm>

#include "constants.h"

using namespace std;

hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG) || defined(CUDA_DEBUG)
	if (result != hipSuccess)
	{
		cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << endl;
		assert(result == hipSuccess);
	}
#endif
	return result;
}

__global__ void gpu_filter(unsigned char* src_img, unsigned char* dst_img, int width) {
	
	uint2 offset = {
		blockIdx.x * THREADS_X + threadIdx.x,
		blockIdx.y * THREADS_Y + threadIdx.y
	};
	
	char neighbors[9];
	neighbors[0] = src_img[offset.y * width + offset.x];
	neighbors[1] = src_img[offset.y * width + offset.x + 1];
	neighbors[2] = src_img[offset.y * width + offset.x + 2];
	neighbors[3] = src_img[(offset.y + 1) * width + offset.x];
	neighbors[4] = src_img[(offset.y + 1) * width + offset.x + 1];
	neighbors[5] = src_img[(offset.y + 1) * width + offset.x + 2];
	neighbors[6] = src_img[(offset.y + 2) * width + offset.x];
	neighbors[7] = src_img[(offset.y + 2) * width + offset.x + 1];
	neighbors[8] = src_img[(offset.y + 2) * width + offset.x + 2];

	char min = neighbors[0];
	for (int i = 1; i < 9; i++) {
		if (neighbors[i] < min)
			min = neighbors[i];
	}

	dst_img[(offset.y + 1) * width + offset.x + 1] = min;
}

__global__ void gpu_filter_lab6(unsigned char* src_img, unsigned char* dst_img, int width) {
	uint2 offset = {
		blockIdx.x * THREADS_X + threadIdx.x,
		blockIdx.y * THREADS_Y + threadIdx.y
	};

	char pixel[3][9];
	for (int color = 0; color < 3; color++) {
		pixel[color][0] = src_img[offset.y * width * 3 + offset.x * 3 + color];
		pixel[color][1] = src_img[offset.y * width * 3 + (offset.x + 1) * 3 + color];
		pixel[color][2] = src_img[offset.y * width * 3 + (offset.x + 2) * 3 + color];
		pixel[color][3] = src_img[(offset.y + 1) * width * 3 + offset.x * 3 + color];
		pixel[color][4] = src_img[(offset.y + 1) * width * 3 + (offset.x + 1) * 3 + color];
		pixel[color][5] = src_img[(offset.y + 1) * width * 3 + (offset.x + 2) * 3 + color];
		pixel[color][6] = src_img[(offset.y + 2) * width * 3 + offset.x * 3 + color];
		pixel[color][7] = src_img[(offset.y + 2) * width * 3 + (offset.x + 1) * 3 + color];
		pixel[color][8] = src_img[(offset.y + 2) * width * 3 + (offset.x + 2) * 3 + color];

		char min = pixel[color][0];
		for (int i = 1; i < 9; i++) {
			if (pixel[color][i] < min)
				min = pixel[color][i];
		}
		dst_img[(offset.y + 1) * width * 3 + (offset.x + 1) * 3 + color] = min;
	}

}