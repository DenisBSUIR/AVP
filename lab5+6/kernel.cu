#include "hip/hip_runtime.h"
#include ""
#include "helper_image.h"

#include <iostream>
#include <vector>
#include <chrono>

#include "cpu_functions.h"
#include "gpu_cores.cuh"
#include "constants.h"

using namespace std;

int main()
{
	vector<char*> filenames(1);
	//filenames[0] = "images/test2.pgm";
	filenames[0] = "images/IMG1_gray.pgm";
	//filenames[1] = "images/IMG2_gray.pgm";
	//filenames[2] = "images/IMG3_gray.pgm";
	//filenames[3] = "images/JohnWick_gray.pgm";
		
	for (int i = 0; i < 1; i++) {
		unsigned char* img = nullptr;
		unsigned int width = 0, height = 0, channels = 0;

		__loadPPM(filenames[i], &img, &width, &height, &channels);

		//cout << filenames[i] << ": " << width << " " << height << " " << channels << endl;

		int mod_width = width + 2;
		int mod_height = height + 2;
		
		//���������� ��������� �� ����� �������
		unsigned char* mod_img = transform_img(img, mod_width, mod_height);

		unsigned char* mod_cpu_result_img = new unsigned char[mod_width * mod_height];

		//����������
		auto start_cpu = chrono::steady_clock::now();
		cpu_filter(mod_img, mod_cpu_result_img, mod_width, mod_height);
		auto end_cpu = chrono::steady_clock::now();
		auto cpu_elapsed = end_cpu - start_cpu;

		//�������� ��������� �� �����
		unsigned char* cpu_result_img = remove_padding(mod_cpu_result_img, width, height, mod_width);


		//���������� ����������� �� GPU

		//������������
		int width_gpu = width, height_gpu = height;

		if (width_gpu % THREADS_X != 0) {
			width_gpu = (width_gpu / THREADS_X + 1) * THREADS_X;
		}

		if (height_gpu % THREADS_Y != 0) {
			height_gpu = (height_gpu / THREADS_Y + 1) * THREADS_Y;
		}

		//������� �����������
		int mod_width_gpu = width_gpu + 2;
		int mod_height_gpu = height_gpu + 2;

		unsigned char* img_for_gpu = prepare_img_for_gpu(img, mod_width_gpu, mod_height_gpu, width, height);

		////���������� � ������� ����
		//unsigned char* gpu_src_img = nullptr;
		//size_t input_pitch = 0;
		//checkCuda(hipMallocPitch(reinterpret_cast<void**>(&gpu_src_img), &input_pitch, mod_width_gpu, mod_height_gpu));
		//checkCuda(hipMemcpy2D(
		//	gpu_src_img,
		//	input_pitch,
		//	img_for_gpu,
		//	mod_width_gpu,
		//	mod_width_gpu,
		//	mod_height_gpu,
		//	hipMemcpyHostToDevice
		//));
	
		//size_t output_pitch = 0;
		//unsigned char* gpu_dst_img = nullptr;
		//hipMallocPitch(reinterpret_cast<void**>(&gpu_dst_img), &output_pitch, mod_width_gpu, mod_height_gpu);

		//float time = 0;
		//hipEvent_t startEvent, stopEvent;
		//checkCuda(hipEventCreate(&startEvent));
		//checkCuda(hipEventCreate(&stopEvent));

		////����
		//dim3 dimGrid(width_gpu / THREADS_X, height_gpu / THREADS_Y);
		//dim3 dimBlock(THREADS_X, THREADS_Y);
		//checkCuda(hipEventRecord(startEvent, 0));
		//gpu_filter <<<dimGrid, dimBlock >>> (gpu_src_img, gpu_dst_img, mod_width_gpu);

		//checkCuda(hipEventRecord(stopEvent, 0));
		//checkCuda(hipEventSynchronize(stopEvent));
		//checkCuda(hipEventElapsedTime(&time, startEvent, stopEvent));

		//unsigned char* mod_gpu_result_img = new unsigned char[mod_width_gpu * mod_height_gpu];
		//checkCuda(hipMemcpy2D(
		//	mod_gpu_result_img,
		//	mod_width_gpu,
		//	gpu_dst_img,
		//	output_pitch,
		//	mod_width_gpu,
		//	mod_height_gpu,
		//	hipMemcpyDeviceToHost
		//));

		//���������� � ������� ����
		unsigned char* gpu_src_img = nullptr;
		size_t input_pitch = 0;
		checkCuda(hipMalloc(&gpu_src_img, mod_width_gpu* mod_height_gpu));
		checkCuda(hipMemcpy(
			gpu_src_img,
			img_for_gpu,			
			mod_width_gpu * mod_height_gpu,
			hipMemcpyHostToDevice
		));

		size_t output_pitch = 0;
		unsigned char* gpu_dst_img = nullptr;
		checkCuda(hipMalloc(&gpu_dst_img, mod_width_gpu * mod_height_gpu));

		float time = 0;
		hipEvent_t startEvent, stopEvent;
		checkCuda(hipEventCreate(&startEvent));
		checkCuda(hipEventCreate(&stopEvent));

		//����
		dim3 dimGrid(width_gpu / THREADS_X, height_gpu / THREADS_Y);
		dim3 dimBlock(THREADS_X, THREADS_Y);
		checkCuda(hipEventRecord(startEvent, 0));
		gpu_filter << <dimGrid, dimBlock >> > (gpu_src_img, gpu_dst_img, mod_width_gpu);

		checkCuda(hipEventRecord(stopEvent, 0));
		checkCuda(hipEventSynchronize(stopEvent));
		checkCuda(hipEventElapsedTime(&time, startEvent, stopEvent));

		unsigned char* mod_gpu_result_img = new unsigned char[mod_width_gpu * mod_height_gpu];
		checkCuda(hipMemcpy(
			mod_gpu_result_img,
			gpu_dst_img,
			mod_width_gpu * mod_height_gpu,
			hipMemcpyDeviceToHost
		));

		unsigned char* gpu_result_img = remove_padding(mod_gpu_result_img, width, height, mod_width_gpu);

		if (!is_equals(cpu_result_img, gpu_result_img, width, height)) {
			cout << "smth went wrong" << endl;
		}

		checkCuda(hipFree(gpu_src_img));
		checkCuda(hipFree(gpu_dst_img));
		checkCuda(hipEventDestroy(startEvent));
		checkCuda(hipEventDestroy(stopEvent));
				
		cout << chrono::duration<double, milli>(cpu_elapsed).count() << " - CPU time" << endl;
		cout << time << " - GPU time" << endl;

		string cpu_result_filename = "CPU_result.pgm";
		string gpu_result_filename = "GPU_result.pgm";
		__savePPM(cpu_result_filename.c_str(), cpu_result_img, width, height, channels);
		__savePPM(gpu_result_filename.c_str(), gpu_result_img, width, height, channels);
	}

	return 0;
}