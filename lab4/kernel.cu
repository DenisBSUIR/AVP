#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <time.h>
#include <chrono>

#include "Functions.h"
#include "Cores.cuh"
#include "Constants.h"

using namespace std;

int main() {
    char* src_matrix, * dst_matrix;

    //��������� ������ �� CPU
    src_matrix = new char [src_matrix_size];
    dst_matrix = new char [dst_matrix_size];

    /*src_matrix = new char* [src_matrix_y];
    for (int i = 0; i < src_matrix_y; i++) {
        src_matrix[i] = new char[src_matrix_x];
    }

    dst_matrix = new char* [dst_matrix_y];
    for (int i = 0; i < dst_matrix_y; i++) {
        dst_matrix[i] = new char[dst_matrix_x];
    }*/


    //�������������
    srand(time(nullptr));
    for (unsigned long long i = 0; i < src_matrix_size; i++) {
        src_matrix[i] = rand() % 10 + '0';
    }

    /*for (int i = 0; i < src_matrix_y; i++) {
        for (int j = 0; j < src_matrix_x; j++) {
            src_matrix[i][j] = rand() % 10 + '0';
        }
    }*/



    //������������� ������� �� CPU
    auto start_cpu = chrono::steady_clock::now();
    cpu_perform_transformation(src_matrix, dst_matrix);
    auto end_cpu = chrono::steady_clock::now();
    auto elapsed_cpu = chrono::duration_cast<chrono::milliseconds>(end_cpu - start_cpu);

    //������������� ������� �� GPU
    size_t size = src_matrix_x * src_matrix_y * sizeof(char);
    char* d_src, * d_dst, *gpu_result_matrix, *gpu_res_matrix_shared;

    gpu_result_matrix = new char[dst_matrix_size];
    gpu_res_matrix_shared = new char[dst_matrix_size];

    int dimBlock_x = 32, dimBlock_y = 32;
    dim3 dimBlock(dimBlock_x, dimBlock_y);
    dim3 dimGrid(blocks_x/dimBlock_x, blocks_y/dimBlock_y);

    checkCuda(hipMalloc(&d_src, size));
    checkCuda(hipMalloc(&d_dst, size));
    checkCuda(hipMemcpy(d_src, src_matrix, size, hipMemcpyHostToDevice));

    hipEvent_t start_event, end_event;
    checkCuda(hipEventCreate(&start_event));
    checkCuda(hipEventCreate(&end_event));
    checkCuda(hipEventRecord(start_event, 0));

    gpu_perform_transformation<<<dimGrid, dimBlock>>>(d_src, d_dst);

    checkCuda(hipEventRecord(end_event, 0));
    checkCuda(hipEventSynchronize(end_event));
    float elapsed_gpu;
    checkCuda(hipEventElapsedTime(&elapsed_gpu, start_event, end_event));
    
    checkCuda(hipMemcpy(gpu_result_matrix, d_dst, size, hipMemcpyDeviceToHost));



    /*checkCuda(hipEventRecord(start_event, 0));
    
    gpu_perform_transformation_shared<<<dimGrid, dimBlock>>>(d_src, d_dst);
    
    checkCuda(hipEventRecord(end_event, 0));
    checkCuda(hipEventSynchronize(end_event));
    float elapsed_gpu_shared;
    checkCuda(hipEventElapsedTime(&elapsed_gpu_shared, start_event, end_event));
    
    checkCuda(hipMemcpy(gpu_res_matrix_shared, d_dst, size, hipMemcpyDeviceToHost));*/


    hipEventDestroy(start_event);
    hipEventDestroy(end_event);
    
    checkCuda(hipFree(d_src));
    checkCuda(hipFree(d_dst));

    if (!check_match(dst_matrix, gpu_result_matrix)) {
        cout << "Wrong algorithm" << endl;
    }

    /*if (!check_match(dst_matrix, gpu_res_matrix_shared)) {
        cout << "Wrong algorithm with shared" << endl;
    }*/

    cout << elapsed_cpu.count() << " - time with CPU" << endl;
    cout << elapsed_gpu << " - time with GPU" << endl;
   // cout << elapsed_gpu_shared << " - time wirh shared GPU" << endl;
    //show_matrix(src_matrix_y, src_matrix_x, src_matrix);
    //cout << endl;
    //show_matrix(dst_matrix_y, dst_matrix_x, dst_matrix);
    //cout << endl;

    /*for (int i = 0; i < src_matrix_y; i++) {
        delete[] src_matrix[i];
    }
    for (int i = 0; i < dst_matrix_y; i++) {
        delete[] dst_matrix[i];
    }*/
    delete[] src_matrix;
    delete[] dst_matrix;
    delete[] gpu_result_matrix;

    system("pause");
    return 0;
}
