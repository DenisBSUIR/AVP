#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <cassert>
#include ""
#include <hip/device_functions.h>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <Windows.h>
#include <hip/hip_runtime.h> 
#include <intrin.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/device_functions.h>


#include "Constants.h"
#include <hip/hip_runtime.h>

using namespace std;

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
//inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG) || defined(CUDA_DEBUG)
	if (result != hipSuccess)
	{
		cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << endl;
		assert(result == hipSuccess);
	}
#endif
	return result;
}

__global__ void gpu_perform_transformation(char* src, char* dst) {
	
	dst[(blockIdx.y * dst_block_y * 32 + threadIdx.y * dst_block_y + 0) * dst_matrix_x + ((blockIdx.x * dst_block_x * 32) + threadIdx.x * dst_block_x)] =
		src[(blockIdx.y * src_block_y * 32 + threadIdx.y * src_block_y) * src_matrix_x + (blockIdx.x * src_block_x * 32 + threadIdx.x * src_block_x)];
	dst[(blockIdx.y * dst_block_y * 32 + threadIdx.y * dst_block_y + 1) * dst_matrix_x + (blockIdx.x * dst_block_x * 32 + threadIdx.x * dst_block_x)] =
		src[(blockIdx.y * src_block_y * 32 + threadIdx.y * src_block_y) * src_matrix_x + (blockIdx.x * src_block_x * 32 + threadIdx.x * src_block_x) + 1];
	dst[(blockIdx.y * dst_block_y * 32 + threadIdx.y * dst_block_y + 2) * dst_matrix_x + (blockIdx.x * dst_block_x * 32 + threadIdx.x * dst_block_x)] =
		src[(blockIdx.y * src_block_y * 32 + threadIdx.y * src_block_y + 1) * src_matrix_x + (blockIdx.x * src_block_x * 32 + threadIdx.x * src_block_x)];
	dst[(blockIdx.y * dst_block_y * 32 + threadIdx.y * dst_block_y + 3) * dst_matrix_x + (blockIdx.x * dst_block_x * 32 + threadIdx.x * dst_block_x)] =
		src[(blockIdx.y * src_block_y * 32 + threadIdx.y * src_block_y + 1) * src_matrix_x + (blockIdx.x * src_block_x * 32 + threadIdx.x * src_block_x) + 1];
}

__global__ void gpu_perform_transformation_shared(char* src, char* dst) {
	__shared__ char smem[64 * 64];
	__shared__ char smem_dst[64 * 64];
	uint2 offset = {
		blockIdx.x * 64 + threadIdx.x,
		blockIdx.y * 64 + threadIdx.y
	};

	smem[threadIdx.y * 64 + threadIdx.x] = src[offset.y * 64 + offset.x];
	smem[threadIdx.y * 64 + threadIdx.x + 32] = src[offset.y * 64 + offset.x + 32];
	smem[(threadIdx.y + 32) * 64 + threadIdx.x] = src[(offset.y + 32) * 64 + offset.x];
	smem[(threadIdx.y + 32) * 64 + threadIdx.x + 32] = src[(offset.y + 32) * 64 + offset.x + 32];

	__syncthreads();

	int a = smem[threadIdx.y * 4 * 64 + threadIdx.x * 2];
	int b = smem[threadIdx.y * 4 * 64 + threadIdx.x * 2 + 1];
	int c = smem[(threadIdx.y * 4 + 1) * 64 + threadIdx.x * 2];
	int d = smem[(threadIdx.y * 4 + 1) * 64 + threadIdx.x * 2 + 1];

	smem_dst[threadIdx.y * 4 * 64 + threadIdx.x * 2] = a;
	smem_dst[(threadIdx.y * 4 + 1) * 64 + threadIdx.x * 2] = b;
	smem_dst[(threadIdx.y * 4 + 2) * 64 + threadIdx.x * 2] = c;
	smem_dst[(threadIdx.y * 4 + 3) * 64 + threadIdx.x * 2] = d;

	dst[offset.y * 64 + offset.x] =	smem_dst[threadIdx.y * 64 + threadIdx.x];
	dst[(offset.y * + 32) * 64 + offset.x] = smem_dst[(threadIdx.y + 32) * 64 + threadIdx.x];
	dst[(offset.y * + 64) * 64 + offset.x] = smem_dst[(threadIdx.y + 64) * 64 + threadIdx.x];
	dst[(offset.y * + 96) * 64 + offset.x] = smem_dst[(threadIdx.y + 96) * 64 + threadIdx.x];
}